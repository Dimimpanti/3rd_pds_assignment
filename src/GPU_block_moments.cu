#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

#define N 10000
#define BLK_THREADS 64
#define SM 6
#define SM_CORES 128
#define ITERATIONS 50
#define MAX_BLK_SIZE 50  

/**
 * Initializes the array and defines its initial uniform random initial state. Our array contains two states either 1 or -1 (atomic "spins").
 * In order to avoid checking for conditions in the parallel section of the program regarding the array limits the array is expanded by 2 on both 
 * dimensions.
 * @param array The pointer to the array that is being initialized
 */
void initializeArray(short int **array){
    srand(time(NULL));

    // i and j start from 1 because the initial array is surrounded by the wrapping rowns and columns
    for (int i = 1; i <= N; i++){
        for (int j = 1; j <= N; j++){

            int rnd = rand() % 1000;  // Get a double random number in (0,1) range
            
            // 0.5 is chosen so that +1 and -1 are 50% each
            if (rnd >= 500){

                // positive spin
                array[i][j] = 1;  
                                
            } else {

                // negatine spin
                array[i][j] = -1;
            }

            // Wrap around for rows: periodic boundary conditions
            if (i == 1){

                // If i == 1 it means that this is the 0 row of the initial array and must be wrapped
                // to the row N - 1 of the final array (see the example above)
                array[N + 1][j] = array[1][j];

            } else if (i == N){

                // If i == N it means that this is the N - 1 row of the initial array and must be wrapped
                // to the row 0 of the final array (see the example above)
                array[0][j] = array[N][j];
            }
                
            // Wrap around for cols: periodic boundary conditions
            if (j == 1){

                // If j == 1 it means that this is the 0 col of the initial array and must be wrapped
                // to the col N - 1 of the final array (see the example above)
                array[i][N + 1] = array[i][1];

            } else if (j == N){

                // If j == N it means that this is the N - 1 col of the initial array and must be wrapped
                // to the col 0 of the final array (see the example above)
                array[i][0] = array[i][N];
            }
        }
    }
}


/**
 * Prints an 1D array like a 2D array for every size + 2 elements
 * 
 * @param array   The array to print
 * @param size  The size of the single row without wrapping
 */
__device__ void printDeviceArray(short int *array, int size){

    for (int i = 0; i < size + 2; i++) {
        for(int j = 0; j < size + 2; j++){

            if (array[i * (size + 2) + j] < 0){
                printf("%hd ", array[i * (size + 2) + j]);
        
            } else {
                printf(" %hd ", array[i * (size + 2) + j]);
            }
        }
        printf("\n");
    }
    printf("\n");
}


/**
 * Finds the sum of all the elements of an 1D array
 * 
 * @param array   The array to find the sum
 * @param size  The size of the single row of the 2D array without wrapping
 * 
 * @return The sum of all the elements of the array
 */ 
__device__ int summation(short int *array, int size){
    
    int sum = 0;
    
    for (int i = 0; i < (size + 2) * (size + 2); i++){
        sum += array[i];
    }

    return sum;
}


/**
 * Sign function implementation
 * 
 * @param sum The sum to find the sign
 * 
 * @return 1 if the sun is greater than 0 else -1
 * 
 */
__device__ short int sign(short int sum){
    return sum > 0 ? 1 : -1;
}


/**
 * @brief Kernel function that simulates the ising model. Every thread calculates more than one moments.
 * 
 * @param arraySize        The size of the global memory array (no wrappings)
 * @param sharedArraySize  The size of the shared memory array (WITH wrappings)
 * @param d_read           The global memory array to read from
 * @param d_write          The global memory array to write to
 * @param blocksPerRow     The number of blocks opend for one side of the array (total number of blocks is this values squared)
 * @param momentsPerThread The number of moments every thread will calculate
 */
__global__ void simulateIsing(int arraySize, int sharedArraySize, short int *d_read, short int *d_write, int blocksPerRow, int momentsPerThread){

    // i and j are the indexes of the first element of each sub block on the global array.
    int i = (blockIdx.x / blocksPerRow) * (sharedArraySize - 2);
    int j = (blockIdx.x % blocksPerRow) * (sharedArraySize - 2);

    // Acount for subarrays overlapping
    j = (j + sharedArraySize > arraySize) ? arraySize + 2 - sharedArraySize : j;

    i = (i + sharedArraySize > arraySize) ? arraySize + 2 - sharedArraySize : i;

    // Index for the flatted out 2D array. This index is the index of the firste moment of the block on the full size array
    int globalIndex = (arraySize + 2) * i + j;
    
    // if (threadIdx.x == 0) {
    //     printf("block id %d, i %d, j %d, globalIndex %d\n", blockIdx.x, i, j, globalIndex);
    // }

    int stride = blockDim.x;

    // From this point forward the problem to be solved is that of the small sub array. 
    // To convert this problem to something previously solve the wrapping lines are subtracted
    // from the size because they are acounted for on the formulas below
    sharedArraySize -= 2;

    for (int k = 0; k < blockDim.x * momentsPerThread; k = k + stride) {
        
        // This is the unique value that every thread in every block gets. 
        // This index does not take in to acount the wrapping lines (See initializeArray function for the wrapping system)
        int index_2d = threadIdx.x + k;

        /**
        * Index for the flatted out 2D array. The array passed in the GPU memory is 1D and already has the wrappings included.
        * To find the corespondance of the 2D index to the 1D index we do the following. The first size + 3 (size = N) elements of the array
        * are elements from the wrapping rows and columns. After that the index_2d is added and finaly index_2d / size calculates the line of the
        * element on the 2D array. For every line we need to add 2 because the first and last element of the array are there for wrapping
        */
        int index = globalIndex + arraySize + 3 + index_2d + (index_2d / sharedArraySize) * (2 + arraySize - sharedArraySize);
        
        // printf("block id %d, t_id %d , index %d, globalIndex %d\n", blockIdx.x, threadIdx.x, index, globalIndex);
        
        if (index <= globalIndex + sharedArraySize * (arraySize + 2) + sharedArraySize){
            
            // printf("blockId: %d, threadId: %d, index: %d, global index: %d\n", blockIdx.x, threadIdx.x, index, newIndex);

            // The sum of all the neighbors read from the sharedArray
            int sum = d_read[index - 1] + d_read[index + 1] + d_read[index - (sharedArraySize + 2)] + d_read[index + (sharedArraySize + 2)] + d_read[index]; 

            // Write the elemets on the write array.
            d_write[index] = sign(sum);

        } else {
            break;
        }
    }
}


/**
 * @brief This function calculates and updates the outer rows and columns that contain the oposite side elements so that the wrapping 
 * can be performed.
 * 
 * @param d_write The just written array to update the wraps
 * @param size The size of the initial array (without the wraps)
 */
__global__ void completeWrapping(short int *d_write, int size){
    
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(j <= size){

        // Update the wrapping rows...
        d_write[size * (size + 2) + size + 2 + j] = d_write[size + 2 + j];  // This formula transforms 2D coordinates to 1D
        d_write[size * (size + 2) + j - size * (size + 2)] = d_write[size * (size + 2) + j];  // This formula transforms 2D coordinates to 1D
        
        
        // ... and columns as well
        d_write[j * (size + 2) + 1 + size] = d_write[j * (size + 2) + 1];  // This formula transforms 2D coordinates to 1D
        d_write[j * (size + 2)] = d_write[j * (size + 2) + size];  // This formula transforms 2D coordinates to 1D
    }
}

__global__ void debugPrints(short int *arr, int size){
    printDeviceArray(arr, size);
}


/**
 * @brief Sums in parallel all the elements of the array. To perform the operation the function must be called twice
 * The first time the array is spit in blocks and the sum of every block is calculated. The sum of every block is then returned
 * in the dout array. The second time of the function is called with the dout as input. The sum of the dout array is returned in the 
 * dout[0]
 * 
 * @param d_out The array to return the results
 * @param arr The input array
 * @param arr_size The size of the inputu array
 */
__global__ void detectStableState(short int *d_out, short int *arr, int arr_size){
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockDim.x;

    const int gridSize = blockDim.x * gridDim.x;
    
    int sum = 0;
    
    for (int i = gthIdx; i < arr_size; i += gridSize)
        sum += arr[i];
    
    __shared__ int shArr[BLK_THREADS];
    shArr[thIdx] = sum;

    __syncthreads();
    
    
    for (int size = blockDim.x / 2; size > 0; size /= 2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];

        __syncthreads();
    }

    if (thIdx == 0)
        d_out[blockIdx.x] = shArr[0];
    
}


/**
 * @brief Finds the optimal value for the sub block size for minimum overlapping
 * 
 * @param size   The array size
 * @param block  The targeted block size
 * @return       The best size for the block of moments (The one that produces less overlapping)
 */
int optimalBlockSize(int size, int block){
    int fit = block;
    
    int initialBlock = block;
    
    int optimalFit = fit;
    int optimalBlock = block;
    
    while (fit != 0 && block >= initialBlock - 20){
        
        fit = block - size % block;
        
        if (fit == block){
            fit = 0;
        }
        
        if (fit < optimalFit){
            optimalFit = fit;
            optimalBlock = block;
        }
        
        // printf("size: %d, block: %d, fit: %d\n", size, block, fit);
        
        block -= 1;
    }
    
    return optimalBlock;
}


int calculateGrid(int *nSimBlocks, int *momentsPerThread, int *blockSize, int arraySize){

    // This is the minimum number of blocks the GPU needs to be 100% utilized. The formula is (Cores per SM div Block size) * Number of SMs 
    int minBlocks = SM_CORES / BLK_THREADS * SM;
    int sharedArraySize;

    // This is the number of the maximum number of moments a block can have based on the amount of shared memory
    int maxMomentsPerBlock = MAX_BLK_SIZE * MAX_BLK_SIZE;

    // This is the minimum total number of moments needed to utilize fully both the shared memory and the CUDA cores of the GPU
    int minTotalMoments = minBlocks * maxMomentsPerBlock;

    // For arrays bigger that minTotalMoments we calculate the block size based on the MAX_BLK_SIZE. This ensures maximum shared memory efficiency
    // since we need to open more blocks than the GPU can run at once.
    if (arraySize * arraySize >= minTotalMoments){
        
        sharedArraySize = optimalBlockSize(arraySize, MAX_BLK_SIZE);  // The best block size for minimal overlap. This is the size without the wrapping lines

        // If the array size is not divided exactly with the block size the last blocks will overlap.
        // This will result on some moments being calculated twice but the number of them will be very small
        // compared with the total moments given a big array.
        *nSimBlocks = arraySize % sharedArraySize == 0 ? arraySize / sharedArraySize : arraySize / sharedArraySize + 1;

        // *nSimBlocks = (*nSimBlocks) * (*nSimBlocks);

        // The number of moments each thread will calculate. The number of threads per blocks is constant
        *momentsPerThread = (sharedArraySize * sharedArraySize) % BLK_THREADS == 0 ? (sharedArraySize * sharedArraySize) / BLK_THREADS : (sharedArraySize * sharedArraySize) / BLK_THREADS + 1;

        *blockSize = BLK_THREADS;

        sharedArraySize += 2;  // Add 2 to the sharedArraySize so that it contains the wrapping lines

    } else {
        // If the total number of moments is less than the minimum we will not need all of the shared memory.
        // In this case we will open the minimum number of blocks that can utilize the GPU fully in terms of cores
        
        // The targeted sharedArraySize
        sharedArraySize = arraySize % 8 == 0 ? arraySize / 8 : arraySize / 8 + 1;

        if (sharedArraySize >= 1){
        
            // If the array size is not divided exactly with the block size the last blocks will overlap.
            // This will result on some moments beign calculated twice but the number of them will be very small
            // compared with the total moments given a big array.
            *nSimBlocks = 8;

            // The number of moments each thread will calculate. The number of threads per blocks is constant
            *momentsPerThread = (sharedArraySize * sharedArraySize) % BLK_THREADS == 0 ? (sharedArraySize * sharedArraySize) / BLK_THREADS : (sharedArraySize * sharedArraySize) / BLK_THREADS + 1;

            
            *blockSize = sharedArraySize * sharedArraySize;
        }

        sharedArraySize += 2;  // Add 2 to the sharedArraySize so that it contains the wrapping lines
    }

    return sharedArraySize;
}


int main(int argc, char **argv){

    // The array is N + 2 size for the wrapping around on both dimensions.
    short int **array1 = (short int **) calloc ((N + 2), sizeof(short int *));

    for (int i = 0; i < N + 2; i++){
        array1[i] = (short int *) calloc ((N + 2), sizeof(short int));
    }

    // Device memory pointers
    short int *d_array1;
    short int *d_array2;


    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    printf("\nStarting simulation \n\n");

    hipEventRecord(start, 0);


    // Allocate the memory for the device arrays
    hipMalloc((void**)&d_array1, sizeof(short int) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_array2, sizeof(short int) * (N + 2) * (N + 2));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Memory allocations time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    // Initialize the array 1 with random -1 and 1 values (50% distribution)
    initializeArray(array1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Initialization time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    // Copy the host memory to the device memory. This transfer also converts the host 2D array to 1D for the device
    for (int i = 0; i < N + 2; i++) {
        hipMemcpy(d_array1 + i * (N + 2), array1[i], sizeof(short int) * (N + 2), hipMemcpyHostToDevice);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Host -> Device time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    int stabilityBlocks = (N * N % BLK_THREADS) ? (N * N / BLK_THREADS + 1) : N * N / BLK_THREADS;

    // Unified memory pointer for detecting stable state
    int *stable_state;
    hipMallocManaged((void **) &stable_state, 3 * sizeof(int));  // Allocate pointer for device and host access (unified memory)

    // Initialize the stable state array with INT_MAX
    stable_state[0] = INT_MAX;
    stable_state[1] = INT_MAX - 1;
    stable_state[2] = INT_MAX - 2;

    short int* dev_out;
    hipMallocManaged((void **)&dev_out, sizeof(short int) * stabilityBlocks);

    int nSimBlocks;
    int momentsPerThread;
    int blockSize;
    
    int sharedArraySize = calculateGrid(&nSimBlocks, &momentsPerThread, &blockSize, N);

    printf("Number of blocks: %d, block size: %d, moments per thread: %d, shared array size: %d\n\n", nSimBlocks * nSimBlocks, blockSize, momentsPerThread, sharedArraySize);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Memory allocations time:  %3.1f ms \n\n", time);
    
    hipEventRecord(start, 0);

    for (int iteration = 0; iteration < ITERATIONS; iteration++){

        // Call the kernel with numberOfBlocks blocks and N_threads. This call introduces a restriction on the size of the array
        // The max number of threads per block is 1024 so the max N is theoretically 32 (practically 30 because of the wrappings)
        simulateIsing <<< nSimBlocks * nSimBlocks, blockSize >>> (N,  sharedArraySize, d_array1, d_array2, nSimBlocks, momentsPerThread);

        hipDeviceSynchronize();

        int wrappingBlocks = (N % BLK_THREADS) ? (N / BLK_THREADS + 1) : N / BLK_THREADS;

        completeWrapping <<<wrappingBlocks, BLK_THREADS>>> (d_array2, N);
        hipDeviceSynchronize();

        // Swap the two arrays.
        short int *tmp = d_array2;
        d_array2 = d_array1;
        d_array1 = tmp;
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("GPU run time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    // Copy the device memory back to host again converting from 1D device array to 2D host array
    for (int i = 0; i < N + 2; i++) {
        hipMemcpy(array1[i], d_array1 + i * (N + 2), sizeof(short int) * (N + 2), hipMemcpyDeviceToHost);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("MEmory Device -> Host time:  %3.1f ms \n\n", time);

    // free memory
    for (int i = 0; i < N + 2; i++){
        free(array1[i]);
    }

    free(array1);
    
    hipFree(d_array1);
    hipFree(d_array2);

    return 0;
}