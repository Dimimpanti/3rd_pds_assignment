#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000
#define BLK_THREADS 32
#define SM 6
#define SM_CORES 128
#define ITERATIONS 50
#define MAX_BLK_SIZE 70  


/**
 * Initializes the array and defines its initial uniform random initial state. Our array contains two states either 1 or -1 (atomic "spins").
 * In order to avoid checking for conditions in the parallel section of the program regarding the array limits the array is expanded by 2 on both 
 * dimensions.
 * @param array The pointer to the array that is being initialized
 */
void initializeArray(short int **array){
    srand(time(NULL));

    // i and j start from 1 because the initial array is surrounded by the wrapping rowns and columns
    for (int i = 1; i <= N; i++){
        for (int j = 1; j <= N; j++){

            int rnd = rand() % 1000;  // Get a double random number in (0,1) range
            
            // 0.5 is chosen so that +1 and -1 are 50% each
            if (rnd >= 500){

                // positive spin
                array[i][j] = 1;  
                                
            } else {

                // negatine spin
                array[i][j] = -1;
            }

            // Wrap around for rows: periodic boundary conditions
            if (i == 1){

                // If i == 1 it means that this is the 0 row of the initial array and must be wrapped
                // to the row N - 1 of the final array (see the example above)
                array[N + 1][j] = array[1][j];

            } else if (i == N){

                // If i == N it means that this is the N - 1 row of the initial array and must be wrapped
                // to the row 0 of the final array (see the example above)
                array[0][j] = array[N][j];
            }
                
            // Wrap around for cols: periodic boundary conditions
            if (j == 1){

                // If j == 1 it means that this is the 0 col of the initial array and must be wrapped
                // to the col N - 1 of the final array (see the example above)
                array[i][N + 1] = array[i][1];

            } else if (j == N){

                // If j == N it means that this is the N - 1 col of the initial array and must be wrapped
                // to the col 0 of the final array (see the example above)
                array[i][0] = array[i][N];
            }
        }
    }
}


/**
 * Prints an 1D array like a 2D array for every size + 2 elements
 * 
 * @param array   The array to print
 * @param size  The size of the single row without wrapping
 */
__device__ void printDeviceArray(short int *array, int size){

    for (int i = 0; i < size + 2; i++) {
        for(int j = 0; j < size + 2; j++){

            if (array[i * (size + 2) + j] < 0){
                printf("%hd ", array[i * (size + 2) + j]);
        
            } else {
                printf(" %hd ", array[i * (size + 2) + j]);
            }
        }
        printf("\n");
    }
    printf("\n");
}

/**
 * Sign function implementation
 * 
 * @param sum The sum to find the sign
 * 
 * @return 1 if the sun is greater than 0 else -1
 * 
 */
__device__ short int sign(short int sum){
    return sum > 0 ? 1 : -1;
}

/**
 * @brief Kernel function that simulates the ising model. Every thread calculates more than one moments using shared memory.
 * 
 * @param arraySize        The size of the global memory array (no wrappings)
 * @param sharedArraySize  The size of the shared memory array (WITH wrappings)
 * @param d_read           The global memory array to read from
 * @param d_write          The global memory array to write to
 * @param blocksPerRow     The number of blocks opend for one side of the array (total number of blocks is this values squared)
 * @param momentsPerThread The number of moments every thread will calculate
 */
__global__ void simulateIsing(int arraySize, int sharedArraySize, short int *d_read, short int *d_write, int blocksPerRow, int momentsPerThread){

    // The size of the shared memory is depended on the size of the sub array.
    __shared__ short int sharedArray[(MAX_BLK_SIZE + 2) * (MAX_BLK_SIZE + 2)];

    // i and j are the indexes of the first element of each sub block on the global array.
    int i = (blockIdx.x / blocksPerRow) * (sharedArraySize - 2);
    int j = (blockIdx.x % blocksPerRow) * (sharedArraySize - 2);


    // Acount for subarrays overlapping
    j = (j + sharedArraySize > arraySize) ? arraySize + 2 - sharedArraySize : j;

    i = (i + sharedArraySize > arraySize) ? arraySize + 2 - sharedArraySize : i;

    // Index for the flatted out 2D array. This index is the index of the firste moment of the block on the global memory
    int globalIndex = (arraySize + 2) * i + j;
    

    // Copy all the elements from the global memory to the shared memory
    for (int x = 0; x < sharedArraySize; x += blockDim.x){

        // Some of the threads at the end of the array go out of bounds.
        // This is because the number of the sub array is not always divided with the number of threads
        // The extra threads simply exit the loop and wait on the syncronise
        if (threadIdx.x + x < sharedArraySize){
            // The index to the shared memory
            int sharedOffset = (threadIdx.x + x) * sharedArraySize;

            // The index to the global memory
            int globalOffset = globalIndex + (threadIdx.x + x) * (arraySize + 2);


            // Every thread copies a row of momnets
            for (int k = 0; k < sharedArraySize; k++) {
                sharedArray[sharedOffset + k] = d_read[globalOffset + k];
            }
        }
    }

    __syncthreads();


    int stride = blockDim.x;
    
    // From this point forward the problem to be solved is that of the small sub array. 
    // To convert this problem to something previously solve the wrapping lines are subtracted
    // from the size because they are acounted for on the formulas below
    sharedArraySize -= 2;

    for (int k = 0; k < blockDim.x * momentsPerThread; k = k + stride) {

        // This is the unique value that every thread in every block gets. 
        // This index does not take in to acount the wrapping lines (See initializeArray function for the wrapping system)
        int index_2d = threadIdx.x + k;

        /**
        * Index for the flatted out 2D array. The array passed in the GPU memory is 1D and already has the wrappings included.
        * To find the corespondance of the 2D index to the 1D index we do the following. The first size + 3 (size = N) elements of the array
        * are elements from the wrapping rows and columns. After that the index_2d is added and finaly index_2d / size calculates the line of the
        * element on the 2D array. For every line we need to add 2 because the first and last element of the array are there for wrapping
        */
        int index = sharedArraySize + 3 + index_2d + (index_2d / sharedArraySize) * 2;

        // if (blockIdx.x == 0){
        //         printf("blockId: %d, threadId: %d, index: %d, global index: %d\n", blockIdx.x, threadIdx.x, index, globalIndex);
        // }

        if (index <= (sharedArraySize + 1) * (sharedArraySize  + 2) - 2){
            
            // printf("block id %d, t_id %d , index %d\n", blockIdx.x, threadIdx.x, index);
            // if (blockIdx.x == 0){
            //     printf("blockId: %d, threadId: %d, index: %d, global index: %d\n", blockIdx.x, threadIdx.x, index, newIndex);
            // }


            // The sum of all the neighbors read from the sharedArray
            int sum = sharedArray[index - 1] + sharedArray[index + 1] + sharedArray[index - (sharedArraySize + 2)] + sharedArray[index + (sharedArraySize + 2)] + sharedArray[index];

            // The index that connects the local memory to the global memory
            int newIndex = globalIndex + arraySize + 3 + index_2d + (index_2d / sharedArraySize) * (2 + arraySize - sharedArraySize); 

            // Write the elemets on the global memory. There is no need to transfer the write array to the Shared memory
            // because only one write access is performed per moment
            d_write[newIndex] = sign(sum);

        } else {
            // If a thread goes out of bounds brakes from the loop
            break;
        }
    }

}


/**
 * @brief This function calculates and updates the outer rows and columns that contain the oposite side elements so that the wrapping 
 * can be performed.
 * 
 * @param d_write The just written array to update the wraps
 * @param size The size of the initial array (without the wraps)
 */
__global__ void completeWrapping(short int *d_write, int size){
    
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if(j <= size){

        // Update the wrapping rows...
        d_write[size * (size + 2) + size + 2 + j] = d_write[size + 2 + j];  // This formula transforms 2D coordinates to 1D
        d_write[size * (size + 2) + j - size * (size + 2)] = d_write[size * (size + 2) + j];  // This formula transforms 2D coordinates to 1D
        
        
        // ... and columns as well
        d_write[j * (size + 2) + 1 + size] = d_write[j * (size + 2) + 1];  // This formula transforms 2D coordinates to 1D
        d_write[j * (size + 2)] = d_write[j * (size + 2) + size];  // This formula transforms 2D coordinates to 1D
    }
}


__global__ void debugPrints(short int *arr, int size){
    printDeviceArray(arr, size);
}


/**
 * @brief Sums in parallel all the elements of the array.
 * @param d_out The array to return the results
 * @param arr The input array
 * @param arr_size The size of the inputu array
 */
__global__ void detectStableState(short int *d_out, short int *arr, int arr_size){
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockDim.x;

    const int gridSize = blockDim.x * gridDim.x;
    
    int sum = 0;
    
    for (int i = gthIdx; i < arr_size; i += gridSize)
        sum += arr[i];
    
    __shared__ int shArr[BLK_THREADS];
    shArr[thIdx] = sum;

    __syncthreads();
    
    
    for (int size = blockDim.x / 2; size > 0; size /= 2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];

        __syncthreads();
    }

    if (thIdx == 0)
        d_out[blockIdx.x] = shArr[0];
    
}


/**
 * @brief Finds the optimal value for the sub block size for minimum overlapping
 * 
 * @param size   The array size
 * @param block  The targeted block size
 * @return       The best size for the block of moments (The one that produces less overlapping)
 */
int optimalBlockSize(int size, int block){
    int fit = block;
    
    int initialBlock = block;
    
    int optimalFit = fit;
    int optimalBlock = block;
    
    while (fit != 0 && block >= initialBlock - 20){
        
        fit = block - size % block;
        
        if (fit == block){
            fit = 0;
        }
        
        if (fit < optimalFit){
            optimalFit = fit;
            optimalBlock = block;
        }
        
        // printf("size: %d, block: %d, fit: %d\n", size, block, fit);
        
        block -= 1;
    }
    
    return optimalBlock;
}


int calculateGrid(int *nSimBlocks, int *momentsPerThread, int *blockSize, int arraySize){

    // This is the minimum number of blocks the GPU needs to be 100% utilized. The formula is (Cores per SM div Block size) * Number of SMs 
    int minBlocks = SM_CORES / BLK_THREADS * SM;
    int sharedArraySize;

    // This is the number of the maximum number of moments a block can have based on the amount of shared memory
    int maxMomentsPerBlock = MAX_BLK_SIZE * MAX_BLK_SIZE;

    // This is the minimum total number of moments needed to utilize fully both the shared memory and the CUDA cores of the GPU
    int minTotalMoments = minBlocks * maxMomentsPerBlock;

    // For arrays bigger that minTotalMoments we calculate the block size based on the MAX_BLK_SIZE. This ensures maximum shared memory efficiency
    // since we need to open more blocks than the GPU can run at once.
    if (arraySize * arraySize >= minTotalMoments){
        
        sharedArraySize = optimalBlockSize(arraySize, MAX_BLK_SIZE);  // The best block size for minimal overlap. This is the size without the wrapping lines

        // If the array size is not divided exactly with the block size the last blocks will overlap.
        // This will result on some moments being calculated twice but the number of them will be very small
        // compared with the total moments (less that 1.8%) given a big array.
        *nSimBlocks = arraySize % sharedArraySize == 0 ? arraySize / sharedArraySize : arraySize / sharedArraySize + 1;

        // The number of moments each thread will calculate. The number of threads per blocks is constant
        *momentsPerThread = (sharedArraySize * sharedArraySize) % BLK_THREADS == 0 ? (sharedArraySize * sharedArraySize) / BLK_THREADS : (sharedArraySize * sharedArraySize) / BLK_THREADS + 1;

        *blockSize = BLK_THREADS;

        sharedArraySize += 2;  // Add 2 to the sharedArraySize so that it contains the wrapping lines

    } else {
        // If the total number of moments is less than the minimum we will not need all of the shared memory.
        // In this case we will open the minimum number of blocks that can utilize the GPU fully in terms of cores
        
        // The targeted sharedArraySize
        sharedArraySize = arraySize % 8 == 0 ? arraySize / 8 : arraySize / 8 + 1;

        if (sharedArraySize >= 1){
        
            // If the array size is not divided exactly with the block size the last blocks will overlap.
            // This will result on some moments beign calculated twice but the number of them will be very small
            // compared with the total moments (less that 1.8%) given a big array.
            *nSimBlocks = 8;

            // The number of moments each thread will calculate. The number of threads per blocks is constant
            *momentsPerThread = (sharedArraySize * sharedArraySize) % BLK_THREADS == 0 ? (sharedArraySize * sharedArraySize) / BLK_THREADS : (sharedArraySize * sharedArraySize) / BLK_THREADS + 1;

            
            *blockSize = sharedArraySize * sharedArraySize;
        }

        sharedArraySize += 2;  // Add 2 to the sharedArraySize so that it contains the wrapping lines
    }

    return sharedArraySize;
}


int main(int argc, char **argv){

    // The array is N + 2 size for the wrapping around on both dimensions.
    short int **array1 = (short int **) calloc ((N + 2), sizeof(short int *));

    for (int i = 0; i < N + 2; i++){
        array1[i] = (short int *) calloc ((N + 2), sizeof(short int));
    }

    // Device memory pointers
    short int *d_array1;
    short int *d_array2;

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("\nStarting simulation \n\n");

    hipEventRecord(start, 0);

    // Allocate the memory for the device arrays
    hipMalloc((void**)&d_array1, sizeof(short int) * (N + 2) * (N + 2));
    hipMalloc((void**)&d_array2, sizeof(short int) * (N + 2) * (N + 2));


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Memory allocations time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    // Initialize the array 1 with random -1 and 1 values (50% distribution)
    initializeArray(array1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Initialization time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    // Copy the host memory to the device memory. This transfer also converts the host 2D array to 1D for the device
    for (int i = 0; i < N + 2; i++) {
        hipMemcpy(d_array1 + i * (N + 2), array1[i], sizeof(short int) * (N + 2), hipMemcpyHostToDevice);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Host -> Device time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);

    int stabilityBlocks = (N * N % BLK_THREADS) ? (N * N / BLK_THREADS + 1) : N * N / BLK_THREADS;

    // Unified memory pointer for detecting stable state
    int *stable_state;
    hipMallocManaged((void **) &stable_state, 3 * sizeof(int));  // Allocate pointer for device and host access (unified memory)

    // Initialize the stable state array with INT_MAX
    stable_state[0] = INT_MAX;
    stable_state[1] = INT_MAX - 1;
    stable_state[2] = INT_MAX - 2;

    short int* dev_out;
    hipMallocManaged((void **)&dev_out, sizeof(short int) * stabilityBlocks);

    int nSimBlocks;
    int momentsPerThread;
    int blockSize;
    
    int sharedArraySize = calculateGrid(&nSimBlocks, &momentsPerThread, &blockSize, N);

    printf("Number of blocks: %d, moments per thread: %d, shared array size: %d, block size %d\n\n", nSimBlocks * nSimBlocks, momentsPerThread, sharedArraySize, blockSize);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Memory allocations time:  %3.1f ms \n\n", time);
    
    hipEventRecord(start, 0);

    for (int iteration = 0; iteration < ITERATIONS; iteration++){

        // Call the kernel with numberOfBlocks blocks and N_threads. This call introduces a restriction on the size of the array
        // The max number of threads per block is 1024 so the max N is theoretically 32 (practically 30 because of the wrappings)
        simulateIsing <<< nSimBlocks * nSimBlocks, blockSize >>> (N,  sharedArraySize, d_array1, d_array2, nSimBlocks, momentsPerThread);  //sharedArraySize * sharedArraySize * sizeof(short int)

        hipDeviceSynchronize();

        int wrappingBlocks = (N % BLK_THREADS) ? (N / BLK_THREADS + 1) : N / BLK_THREADS;

        completeWrapping <<<wrappingBlocks, BLK_THREADS>>> (d_array2, N);
        hipDeviceSynchronize();


        // Swap the two arrays.
        short int *tmp = d_array2;
        d_array2 = d_array1;
        d_array1 = tmp;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("GPU run time:  %3.1f ms \n\n", time);

    hipEventRecord(start, 0);
    
    // Copy the device memory back to host again converting from 1D device array to 2D host array
    for (int i = 0; i < N + 2; i++) {
        hipMemcpy(array1[i], d_array1 + i * (N + 2), sizeof(short int) * (N + 2), hipMemcpyDeviceToHost);
        // hipMemcpy(array2[i], d_array2 + i * (N + 2), sizeof(short int) * (N + 2), hipMemcpyDeviceToHost);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("MEmory Device -> Host time:  %3.1f ms \n\n", time);

    // free memory
    for (int i = 0; i < N + 2; i++){
        free(array1[i]);
    }

    free(array1);
    
    hipFree(d_array1);
    hipFree(d_array2);

    return 0;
}

